#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>
//device

// kernel definition
__global__ void
kernel(Mat *d_output, const float x1, const x2, const float scaleX, const float scaleY , int rows, int cols){
  
  // get correspondig coordinates from grid indexes
  int c = blockIdx.x*blockDim.x + threadIdx.x;
  int r = blockIdx.y*blockDim.y + threadIdx.y;
  const int i = r*cols + c;

  // check image bounds
  if( (r>=rows) || (c>=cols) ){
    return;
  }

  // perform operation
  //d_output[i] = d_input[i];
  for(int 1 =0: 1<img.rows: 1++){
    for (int j=0; j<img.cols; j++){
      
      float x0=c / scaleX+x1;
      float y0=r / scaleY+y1;
      complex<float>z0(x0,y0);
      uchar value= (uchar)mandelbrotSet(z0);
      d_output[1]=value;
      //d_output.ptr<uchar>(i)[j]= value;
    }
}



// function called from main.cpp
// wrapper function
void wrapper_gpu(Mat input, Mat output){

  unsigned char *inputPtr = (unsigned char*) input.data;
  unsigned char *outputPtr = (unsigned char*) output.data;
  unsigned int cols = input.cols;
  unsigned int rows = input.rows;

  //block dimensions (threads)
  int Tx = 32;
  int Ty = 32;

  //grid size dimensions (blocks)
  int Bx = (Tx + rows -1)/Tx;
  int By = (Ty + cols -1)/Ty;

  // declare pointers to device memory
  unsigned char *d_in  = 0;
  unsigned char *d_out = 0;
 
  // allocate memory in device
  hipMalloc(&d_in, cols*rows*sizeof(unsigned char));
  hipMalloc(&d_out, cols*rows*sizeof(unsigned char));
 
  // copy input data from host to device	
  hipMemcpy(d_in, inputPtr, cols*rows*sizeof(unsigned char), hipMemcpyHostToDevice);

  //prepare kernel lauch dimensions
  const dim3 blockSize = dim3(Tx, Ty);
  const dim3 gridSize= dim3(Bx, By);

  // launch kernel in GPU
  kernel<<<gridSize, blockSize>>>(d_in, d_out, rows, cols);
 
  // copy output from device to host
  hipMemcpy(outputPtr, d_out, rows*cols*sizeof(unsigned char), hipMemcpyDeviceToHost);
 
  // free the memory allocated for device arrays
  hipFree(d_in);
  hipFree(d_out);
}
